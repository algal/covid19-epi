
#include <hip/hip_runtime.h>
#define deg2rad(deg) (deg * M_PI / 180)
#define rad2deg(rad) (rad * 180 / M_PI)

static double calc_community_infect(float kappa, float omega, int severe, double d, float betac_scale) {

	/* need to work on this.  Perhaps we take a random distance for each two people based on population density, number of people in county, county area, etc. */
	float zeta[]={0.1, 0.25, 0.5, 0.75, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.75, 0.50, 0.25, 0.25, 0.25} ; //   # Travel related parameter for community transmission. Ferguson Nature 2006
	double fd;
	float betac=0.103 ; // Scaled from betac=0.075 in influenza pandemic with R0=1.6, COVID-19 R0=2.2 (Ferguson 2020)

	fd=1/(1+pow((d/4), 3)); //kernel density function as parameterized for GB.
	return (betac_scale*betac*kappa*fd*(1+severe*(omega-1)));
}

static double distance(double lat1, double lon1, double lat2, double lon2, char unit) {
    double theta, dist;
    if ((lat1 == lat2) && (lon1 == lon2)) {
      return 0;
    }
    else {
      theta = lon1 - lon2;
  //    dist = sin(deg2rad(lat1)) * sin(deg2rad(lat2)) + cos(deg2rad(lat1)) * cos(deg2rad(lat2)) * cos(deg2rad(theta));
      double ang1,ang2;
      ang1 = deg2rad(lat1);
      ang2 = deg2rad(lat2);
      dist = cos(ang1) * cos(ang2) * ( 1.0 + cos(deg2rad(theta)) ) - cos(ang1 + ang2);
      dist = acos(dist);
      dist = rad2deg(dist);
      dist = dist * 60 * 1.1515;
      switch(unit) {
        case 'M':
          break;
        case 'K':
          dist = dist * 1.609344;
          break;
        case 'N':
          dist = dist * 0.8684;
          break;
      }
      return (dist);
    }
}

static float calc_kappa(float t, float tau, int symptomatic, float dt, float * kappa_vals, int hosp, int icu, int full_kappa, float R0_scale) {

	float kappa;
	float t1;
	int t2;
	//###Determine kappa for infected person.  This is the infectiousness of the person based on time since infection started.  Latency period is 4.6 days.  Infection starts at 5.1 days and lasts for 6 days.  Sympotmatic people are twice as likely to infect others as asymptomatic.
	// Kappa is a log normal function with mean of -0.72 and standard deviation of 1.8.  From Ferguson Nature 2005
	if (t-tau <= 4.6) {
		kappa=0.;
	} else if (t-tau>11.1 && hosp==0 && icu==0) {
		kappa=0.; //# Recovered or dead
	} else {
		/* First 2 lines calculates kappa on the fly, second two get precalculated kappa from array. */
                if (full_kappa) {
                    t1=(log(t-tau-4.6)+0.72)/1.8;
                    kappa=exp(-0.5*pow(t1,2))/((t-tau-4.6)*1.8*sqrt(2*M_PI));
                } else {
                    t2=(t-tau)/dt;
                    kappa=kappa_vals[t2];
                }
	}
	if (symptomatic==0) {
		kappa=kappa*0.5;
	}
	return(kappa*R0_scale);
}

extern "C" void locale_infectious_step(int j, int num_infectious, int* infectious, float Ic, int* intervene, float t, float* tau, float* tauI, float* interIc, int* symptomatic, float dt, float* kappa_vals, int* hosp_pop, int* icu_pop, float* lat_locale, float* lon_locale, int* locale_HH, int* HH, struct locale* locale_list, float omega, int* severe, double* out_tmp_comm_inf, int full_kappa, float R0_scale, float betac_scale) {
	double tmp_comm_inf = 0.0;
	int i;
	for (i=0; i<num_infectious; i++) {
		int infec_person; //Counter for infected person.
		float kappa; // #Infectiousness
		float tIc;
		infec_person = infectious[i];
		tIc = Ic;
		if ( intervene[infec_person] > 0 && t>tau[infec_person]+tauI[intervene[infec_person]]) {
			tIc = interIc[intervene[infec_person]];
		}
		kappa = calc_kappa( t,  tau[infec_person], symptomatic[infec_person], dt, kappa_vals, hosp_pop[infec_person], icu_pop[infec_person], full_kappa, R0_scale);
	
		if (hosp_pop[infec_person]==0) {
			float d; //distance between people.
			// Community transmission //
#if !defined(USE_LOCALE_DISTANCE)
			d = distance(lat_locale[j], lon_locale[j], lat_locale[locale_HH[HH[infec_person]]], lon_locale[locale_HH[HH[infec_person]]], 'K');
#else
			d = locale_distance(locale_list[j], locale_list[locale_HH[HH[infec_person]]]);
#endif
			tmp_comm_inf += tIc*calc_community_infect( kappa, omega, severe[infec_person], d, betac_scale);
		}
	}

	*out_tmp_comm_inf = tmp_comm_inf;
}