#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "locale.h"

#include "Remotery.h"


/* Earth flattening and radius according to GRS80 taken from https://en.wikipedia.org/wiki/Geodetic_Reference_System_1980 */
#define FLATTENING 0.003352810681183637418
#define Earth_Radius_GRS80 6378.137

/* Earth Mean Radius according to IUGG, taken from https://en.wikipedia.org/wiki/Earth_radius */
#define Earth_Radius_Mean 6371.0087714

#define x(radius, lat, lon) ((radius)*sin(deg2rad(90.0-(lat)))*cos(deg2rad((lon))))
#define y(radius, lat, lon) ((radius)*sin(deg2rad(90.0-(lat)))*sin(deg2rad((lon))))
#define z(radius, lat) ((radius)*cos(deg2rad(90.0-(lat))))

#undef locale_distance
#if defined(USE_LAMBERT)
#define locale_distance d_locale_distance_Lambert
#else
#define locale_distance d_locale_distance_GCD_1
#endif

/* Distance using Lambert formula, taken from https://en.wikipedia.org/wiki/Geographical_distance */
static __device__ double d_locale_distance_Lambert(struct locale l1, struct locale l2)
{
    double d, ca, sca, cca, p, q, x, y, sp, sq, cp, cq;

    if (l1.lat == l2.lat && l1.lon == l2.lon) {
        return 0.0;
    }
    ca = acos(sin(l1.rlat) * sin(l2.rlat) + cos(l1.rlat) * cos(l2.rlat) * cos(l1.rlon - l2.rlon));
    sca = sin(ca/2);
    cca = cos(ca/2);
    p = (l1.rbeta+l2.rbeta)/2;
    sp = sin(p);
    cp = cos(p);
    q = (l2.rbeta-l1.rbeta)/2;
    sq = sin(q);
    cq = cos(q);
    x = (ca-sin(ca))*(sp*sp*cq*cq/(cca*cca));
    y = (ca+sin(ca))*(cp*cp*sq*sq/(sca*sca));
    d = Earth_Radius_GRS80*(ca - FLATTENING/2*(x+y));
    return d;
}

/* Great Circle Distance */
static __device__ double d_locale_distance_GCD_1(struct locale l1, struct locale l2)
{
    double d;
    if (l1.lat == l2.lat && l1.lon == l2.lon) {
        return 0.0;
    }
    d = (l1.x * l2.x + l1.y * l2.y + l1.z * l2.z) / (Earth_Radius_Mean*Earth_Radius_Mean);
    return acos(d)*Earth_Radius_Mean;
}

#define deg2rad(deg) (deg * M_PI / 180)
#define rad2deg(rad) (rad * 180 / M_PI)

static __device__ double calc_community_infect(float kappa, float omega, int severe, double d, float betac_scale) {

	/* need to work on this.  Perhaps we take a random distance for each two people based on population density, number of people in county, county area, etc. */
	float zeta[]={0.1, 0.25, 0.5, 0.75, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.75, 0.50, 0.25, 0.25, 0.25} ; //   # Travel related parameter for community transmission. Ferguson Nature 2006
	double fd;
	float betac=0.103 ; // Scaled from betac=0.075 in influenza pandemic with R0=1.6, COVID-19 R0=2.2 (Ferguson 2020)

	fd=1/(1+pow((d/4), 3)); //kernel density function as parameterized for GB.
	return (betac_scale*betac*kappa*fd*(1+severe*(omega-1)));
}

static __device__ double d_distance(double lat1, double lon1, double lat2, double lon2, char unit) {
    double theta, dist;
    if ((lat1 == lat2) && (lon1 == lon2)) {
      return 0;
    }
    else {
      theta = lon1 - lon2;
  //    dist = sin(deg2rad(lat1)) * sin(deg2rad(lat2)) + cos(deg2rad(lat1)) * cos(deg2rad(lat2)) * cos(deg2rad(theta));
      double ang1,ang2;
      ang1 = deg2rad(lat1);
      ang2 = deg2rad(lat2);
      dist = cos(ang1) * cos(ang2) * ( 1.0 + cos(deg2rad(theta)) ) - cos(ang1 + ang2);
      dist = acos(dist);
      dist = rad2deg(dist);
      dist = dist * 60 * 1.1515;
      switch(unit) {
        case 'M':
          break;
        case 'K':
          dist = dist * 1.609344;
          break;
        case 'N':
          dist = dist * 0.8684;
          break;
      }
      return (dist);
    }
}

static __device__ float calc_kappa(float t, float tau, int symptomatic, float dt, float const* kappa_vals, int hosp, int icu, int full_kappa, float R0_scale) {

	float kappa;
	float t1;
	int t2;
	//###Determine kappa for infected person.  This is the infectiousness of the person based on time since infection started.  Latency period is 4.6 days.  Infection starts at 5.1 days and lasts for 6 days.  Sympotmatic people are twice as likely to infect others as asymptomatic.
	// Kappa is a log normal function with mean of -0.72 and standard deviation of 1.8.  From Ferguson Nature 2005
	if (t-tau <= 4.6) {
		kappa=0.;
	} else if (t-tau>11.1 && hosp==0 && icu==0) {
		kappa=0.; //# Recovered or dead
	} else {
		/* First 2 lines calculates kappa on the fly, second two get precalculated kappa from array. */
                if (full_kappa) {
                    t1=(log(t-tau-4.6)+0.72)/1.8;
                    kappa=exp(-0.5*pow(t1,2))/((t-tau-4.6)*1.8*sqrt(2*M_PI));
                } else {
                    t2=(t-tau)/dt;
                    kappa=kappa_vals[t2];
                }
	}
	if (symptomatic==0) {
		kappa=kappa*0.5;
	}
	return(kappa*R0_scale);
}

struct LoopInvariantData {
    thrust::device_vector<int> infectious;
    thrust::device_vector<int> intervene;
    thrust::device_vector<float> tau;
    thrust::device_vector<float> tauI;
    thrust::device_vector<float> interIc;
    thrust::device_vector<int> symptomatic;
    thrust::device_vector<float> kappa_vals;
    thrust::device_vector<int> hosp_pop;
    thrust::device_vector<int> icu_pop;
    thrust::device_vector<float> lat_locale;
    thrust::device_vector<float> lon_locale;
    thrust::device_vector<int> locale_HH;
    thrust::device_vector<int> HH;
    thrust::device_vector<struct locale> locale_list;
    thrust::device_vector<int> severe;
};


static __global__ void locale_infectious_step_kernel(
    int j,
    int num_infectious,
    float Ic,
    float t,
    float dt,
    float omega,
    int full_kappa,
    float R0_scale,
    float betac_scale,

    int const* infectious,
    int const* intervene,
    float const* tau,
    float const* tauI,
    float const* interIc,
    int const* symptomatic,
    float const* kappa_vals,
    int const* hosp_pop,
    int const* icu_pop,
    float const* lat_locale,
    float const* lon_locale,
    int const* locale_HH,
    int const* HH,
    struct locale const* locale_list,
    int const* severe,

    double* tmp_comm_inf_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_infectious) {
        double tmp_comm_inf = 0.0;
        int infec_person; //Counter for infected person.
        float kappa; // #Infectiousness
        float tIc;
        infec_person = infectious[i];
        tIc = Ic;
        if ( intervene[infec_person] > 0 && t>tau[infec_person]+tauI[intervene[infec_person]]) {
            tIc = interIc[intervene[infec_person]];
        }
        kappa = calc_kappa( t,  tau[infec_person], symptomatic[infec_person], dt, kappa_vals, hosp_pop[infec_person], icu_pop[infec_person], full_kappa, R0_scale);
    
        if (hosp_pop[infec_person]==0) {
            float d; //distance between people.
            // Community transmission //
#if !defined(USE_LOCALE_DISTANCE)
            d = d_distance(lat_locale[j], lon_locale[j], lat_locale[locale_HH[HH[infec_person]]], lon_locale[locale_HH[HH[infec_person]]], 'K');
#else
            d = locale_distance(locale_list[j], locale_list[locale_HH[HH[infec_person]]]);
#endif
            tmp_comm_inf += tIc * calc_community_infect( kappa, omega, severe[infec_person], d, betac_scale);
        }
    
        tmp_comm_inf_arr[i] = tmp_comm_inf;
    }
}

void locale_infectious_step(LoopInvariantData const& lid, int population, int j, int num_households, int num_infectious, float Ic, float t, float dt, float omega, double& out_tmp_comm_inf, int full_kappa, float R0_scale, float betac_scale) {
    if (num_infectious == 0) {
        out_tmp_comm_inf = 0.0;
        return;
    }
    
    thrust::device_vector<double> d_tmp_comm_inf_arr(num_infectious, 0.0);

    // Run kernel
    size_t const THREAD_COUNT = 512;
    size_t const BLOCK_COUNT = (num_infectious + THREAD_COUNT - 1) / THREAD_COUNT;
    locale_infectious_step_kernel<<<BLOCK_COUNT, THREAD_COUNT>>>(
        j,
        num_infectious,
        Ic,
        t,
        dt,
        omega,
        full_kappa,
        R0_scale,
        betac_scale,
        
        lid.infectious.data().get(),
        lid.intervene.data().get(),
        lid.tau.data().get(),
        lid.tauI.data().get(),
        lid.interIc.data().get(),
        lid.symptomatic.data().get(),
        lid.kappa_vals.data().get(),
        lid.hosp_pop.data().get(),
        lid.icu_pop.data().get(),
        lid.lat_locale.data().get(),
        lid.lon_locale.data().get(),
        lid.locale_HH.data().get(),
        lid.HH.data().get(),
        lid.locale_list.data().get(),
        lid.severe.data().get(),

        d_tmp_comm_inf_arr.data().get());

    out_tmp_comm_inf = thrust::reduce(d_tmp_comm_inf_arr.begin(), d_tmp_comm_inf_arr.end(), 0.0, thrust::plus<double>());
}

extern "C" void locale_infectious_loop(int num_locale, int population, int j, int num_households, int num_infectious, int* infectious, float Ic, int* intervene, float t, float* tau, float* tauI, float* interIc, int* symptomatic, float dt, float* kappa_vals, int count_kappa_vals, int* hosp_pop, int* icu_pop, float* lat_locale, float* lon_locale, int* locale_HH, int* HH, struct locale* locale_list, float omega, int* severe, int full_kappa, float R0_scale, float betac_scale, double* commun_nom1, double* fd_tot) {
    rmt_BeginCPUSample(LocaleLoop, 0);

    size_t const num_I = 9;

    // Allocate device and host arrays
    LoopInvariantData lid;
    lid.infectious.assign(infectious, infectious + num_infectious);
    lid.intervene.assign(intervene, intervene + population);
    lid.tau.assign(tau, tau + population);
    lid.tauI.assign(tauI, tauI + num_I);
    lid.interIc.assign(interIc, interIc + num_I);
    lid.symptomatic.assign(symptomatic, symptomatic + population);
    lid.kappa_vals.assign(kappa_vals, kappa_vals + count_kappa_vals);
    lid.hosp_pop.assign(hosp_pop, hosp_pop + population);
    lid.icu_pop.assign(icu_pop, icu_pop + population);
    lid.lat_locale.assign(lat_locale, lat_locale + num_locale);
    lid.lon_locale.assign(lon_locale, lon_locale + num_locale);
    lid.locale_HH.assign(locale_HH, locale_HH + num_households);
    lid.HH.assign(HH, HH + population);
    lid.locale_list.assign(locale_list, locale_list + num_locale);
    lid.severe.assign(severe, severe + population);

	for (j=0; j<num_locale; j++) {
		double tmp_comm_inf = 0.0;

		rmt_BeginCPUSample(LocaleInfectiousLoop, RMTSF_Aggregate);

		locale_infectious_step(lid, population, j, num_households, num_infectious, Ic, t, dt, omega, tmp_comm_inf, full_kappa, R0_scale, betac_scale);

		commun_nom1[j] = tmp_comm_inf / fd_tot[j];
//		printf("infec_person locale %i j %i num_infectious %i tmp %f actual %f %f \n", num_locale, j, num_infectious, tmp_comm_inf, commun_nom1[j], commun_nom1[j]/fd_tot[j]);
		rmt_EndCPUSample(); // LocaleInfectiousLoop	
	}
	rmt_EndCPUSample(); // LocaleLoop
}